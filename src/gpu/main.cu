#include "hip/hip_runtime.h"
//==============================================================================================
// Originally written in 2016 by Peter Shirley <ptrshrl@gmail.com>
//
// To the extent possible under law, the author(s) have dedicated all copyright and related and
// neighboring rights to this software to the public domain worldwide. This software is
// distributed without any warranty.
//
// You should have received a copy (see file COPYING.txt) of the CC0 Public Domain Dedication
// along with this software. If not, see <http://creativecommons.org/publicdomain/zero/1.0/>.
//==============================================================================================

#include "camera.h"
#include "color.h"
#include "hittable_list.h"
#include "rtweekend.h"
#include "scene.h"

// window
#include "external/window.h"

#include <atomic>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <>
#include <iomanip>
#include <iostream>
#include <thread>

// 一些CUDA辅助函数和宏

// limited version of checkCudaErrors from hip/hip_runtime_api.h in CUDA examples
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)

void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " << file << ":"
                  << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

// ======================================================
// 这里使用一个“全局”函数，来获取每个线程的随机数

// 表示随机数状态的全局数组
__device__ hiprandState *dev_rand_state;
__device__ int          rand_width, rand_height;
__device__ inline float random_float()
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= rand_width || j >= rand_height)
        return 0.0f;

    int   id = j * rand_width + i;
    float x  = hiprand_uniform(&dev_rand_state[id]);

    return x;
}

// ======================================================

// 初始化随机数状态

__global__ void rand_init(int width, int height, hiprandState *rand_state)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= width || j >= height)
        return;

    int id = j * width + i;
    if (id == 0) {
        dev_rand_state = rand_state;
        rand_width     = width;
        rand_height    = height;
    }
    hiprand_init(42 + id, 0, 0, &rand_state[id]);
}

// 构造场景

__global__ void setup_secne(hittable_list **scene_ptr)
{
    scene_ptr[0] = new hittable_list;
    scene_ptr[1] = new hittable_list;
    cornell_box(*scene_ptr[0], *scene_ptr[1]);
}

// 构造场景

__global__ void cleanup_secne(hittable_list **scene_ptr)
{
    delete scene_ptr[0];
    delete scene_ptr[1];
}

// 渲染

__global__ void ray_radiance(color          *fb,
                             int             width,
                             int             height,
                             int             samples_per_pixel,
                             int             max_depth,
                             camera         *cam,
                             hittable_list **scene_ptr)
{
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= width || j >= height)
        return;

    const hittable &world  = *scene_ptr[0];
    const hittable &lights = *scene_ptr[1];

    color          radiance(0, 0, 0);
    hit_record     rec;
    scatter_record srec;
    color          accumL, accumR;
    ray            r;

    for (int s = 0; s < samples_per_pixel; ++s) {
        r      = cam->get_ray(i, j, width, height);
        accumL = color();
        accumR = color(1.0f, 1.0f, 1.0f);

        for (int depth = 0; depth < max_depth; depth++) {
            // If the ray hits nothing, return the background color.
            if (!world.hit(r, 0.001f, infinity, rec)) {
                accumL += accumR * background_radiance(r);
                break;
            }

            accumL += accumR * rec.mat_ptr->emitted(r, rec, rec.u, rec.v, rec.p);

            if (!rec.mat_ptr->scatter(r, rec, srec))
                break;

            accumR = accumR * srec.attenuation;

            if (srec.is_specular) {
                r = srec.specular_ray;
            }
            else {
                hittable_pdf light(lights, rec.p);
                mixture_pdf  p(light, *srec.pdf_ptr);
                ray          scattered = ray(rec.p, p.generate());
                auto         pdf_val   = p.value(scattered.direction());

                accumR *= rec.mat_ptr->scattering_pdf(r, rec, scattered) / pdf_val;
                r = scattered;
            }
        }

        radiance += accumL;
    }

    fb[j * width + i] = radiance;
}

int main()
{
    // Image

    const int  image_width       = 800;
    const int  image_height      = 800;
    const int  thread_width      = 32;
    const int  thread_height     = 16;
    const int  samples_per_pixel = 200;
    const int  max_depth         = 5;
    const auto aspect_ratio      = static_cast<float>(image_width) / image_height;

    // World

    hittable_list **scene_ptr;
    checkCudaErrors(hipMalloc((void **)&scene_ptr, 2 * sizeof(hittable_list *)));

    setup_secne<<<1, 1>>>(scene_ptr);

    // Camera

    point3 lookfrom(278, 278, -800);
    point3 lookat(278, 278, 0);
    vec3   vup(0, 1, 0);
    auto   dist_to_focus = 10.0f;
    auto   aperture      = 0.001f;
    auto   vfov          = 40.0f;
    auto   time0         = 0.0f;
    auto   time1         = 1.0f;

    camera *cam;
    checkCudaErrors(hipMallocManaged((void **)&cam, sizeof(camera)));
    *cam = camera(lookfrom, lookat, vup, vfov, aspect_ratio, aperture, dist_to_focus, time0, time1);

    // Render Init

    Window window(image_width, image_height, "Ray tracing (GPU)");
    size_t num_pixels = image_width * image_height;
    color *frame_buffer;
    checkCudaErrors(hipMallocManaged((void **)&frame_buffer, num_pixels * sizeof(color)));

    hiprandState *rand_state;
    checkCudaErrors(hipMalloc((void **)&rand_state, num_pixels * sizeof(hiprandState)));

    dim3 blocks((image_width + thread_width - 1) / thread_width,
                (image_height + thread_height - 1) / thread_height);
    dim3 threads(thread_width, thread_height);

    rand_init<<<blocks, threads>>>(image_width, image_height, rand_state);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    // Render

    auto start_time = std::chrono::high_resolution_clock::now();

    ray_radiance<<<blocks, threads>>>(frame_buffer,
                                      image_width,
                                      image_height,
                                      samples_per_pixel,
                                      max_depth,
                                      cam,
                                      scene_ptr);
    checkCudaErrors(hipGetLastError());
    checkCudaErrors(hipDeviceSynchronize());

    auto  end_time   = std::chrono::high_resolution_clock::now();
    auto  elapsed_ms = std::chrono::duration_cast<std::chrono::milliseconds>(end_time - start_time);
    float elapsed_seconds = elapsed_ms.count() * 0.001f;
    int64_t total_rays    = (int64_t)samples_per_pixel * image_height * image_width;
    std::cout << std::fixed << std::setprecision(3) << "\nDone after " << elapsed_seconds
              << " seconds, " << total_rays / elapsed_seconds / 1000000.f << " Mrays per second.\n";

    // Copy frame buffer to window
    for (int j = image_height - 1; j >= 0; --j) {
        for (int i = 0; i < image_width; ++i) {
            size_t pixel_index = j * image_width + i;
            color  c           = radiance_to_color(frame_buffer[pixel_index], samples_per_pixel);
            *window(i, image_height - 1 - j) = color_to_rgb_integer(c);
        }
    }

    window.update();
    while (window.is_run()) {
        window.dispatch();
        std::this_thread::sleep_for(std::chrono::milliseconds(10));
    }

    // Cleanup

    cleanup_secne<<<1, 1>>>(scene_ptr);

    checkCudaErrors(hipFree(scene_ptr));
    checkCudaErrors(hipFree(cam));
    checkCudaErrors(hipFree(frame_buffer));
    checkCudaErrors(hipFree(rand_state));
}
